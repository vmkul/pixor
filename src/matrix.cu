#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>
#include "debug.h"
#include "matrix.h"

using namespace Pixor;

const int NUM_BLOCKS = 100;
const int THREADS_PER_BLOCK = 512;

__host__
__device__
Row::Row(int length, double *r) :
  r(r),
  length(length)
  {}

__host__
__device__
double &Row::operator[](int index)
{
  assert(index >= 0 && index < length);
  return r[index];
}


Matrix::Matrix(int width, int height) :
  width(width),
  height(height)
{
  hipMallocManaged(&m, width * height * sizeof(double));

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      (*this)[i][j] = 0;
    }
  }
}


Matrix::Matrix(std::vector<std::vector<double>> matrix) :
  width(matrix[0].size()),
  height(matrix.size())
{
  hipMallocManaged(&m, width * height * sizeof(double));

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      (*this)[i][j] = matrix[i][j];
    }
  }
}

__host__
__device__
int Matrix::get_width()
{
  return width;
}

__host__
__device__
int Matrix::get_height()
{
  return height;
}

__host__
__device__
Row Matrix::operator[](int index)
{
  assert(index >= 0 && index < height);
  return Row(width, m + index * width);
}


void Matrix::print()
{
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      std::cout << (*this)[i][j] << " ";
    }
    std::cout << '\n';
  }
}

__global__
void matrix_power(Matrix src, Matrix dest, int exponent)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    int i = index / width;
    int j = index - i * width;
    dest[i][j] = pow(src[i][j], exponent);
  }
}

Matrix Matrix::power(int exponent)
{
  Matrix res(width, height);

  matrix_power<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, exponent);
  hipDeviceSynchronize();
  
  return res;
}


Matrix Matrix::add(Matrix other)
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res[i][j] = (*this)[i][j] + other[i][j];
    }
  }
  
  return res;
}


Matrix Matrix::mult(float k)
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res[i][j] = (*this)[i][j] * k;
    }
  }
  
  return res;
}


Matrix Matrix::div(float k)
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res[i][j] = (*this)[i][j] / k;
    }
  }
  
  return res;
}


Matrix Matrix::exp()
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res[i][j] = std::exp((*this)[i][j]);
    }
  }
  
  return res;
}


Matrix Matrix::neg() {
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res[i][j] = -(*this)[i][j];
    }
  }
  
  return res;
}


double Matrix::sum() {
  double res = 0;

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res += (*this)[i][j];
    }
  }

  return res;
}


double Matrix::max() {
  double res = (*this)[0][0];

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      double val = (*this)[i][j];
      if (val > res) res = val;
    }
  }

  return res;
}

__global__
void matrix_convolve(Matrix src, Matrix dest, Matrix kernel, int width, int height, int kernel_width, int kernel_height)
{
  int offset = (kernel_width - 1) / 2;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    int row = index / width;
    int col = index - row * width;
    double val = 0;

    for (int kernel_row = 0; kernel_row < kernel_height; kernel_row++) {
      for (int kernel_col = 0; kernel_col < kernel_width; kernel_col++) {
        int src_row = row + kernel_row - offset;
        int src_col = col + kernel_col - offset;
        if (src_row < 0 || src_row > height - 1) {
          src_row = row + (kernel_height - kernel_row) - offset;
        }
        if (src_col < 0 || src_col > width - 1) {
          src_col = col + (kernel_width - kernel_col) - offset;
        }
        double src_val = src[src_row][src_col];
        double k_val = kernel[kernel_height - 1 - kernel_row][kernel_width - 1 - kernel_col];

        val += k_val * src_val;
      }
    }

    dest[row][col] = val;
  }
}


Matrix Matrix::convolve(Matrix kernel) {
  Matrix res(width, height);
  int kernel_width = kernel.get_width();
  int kernel_height = kernel.get_height();
  assert(kernel_width == kernel_height);
  assert(kernel_width % 2 == 1);

  matrix_convolve<<<100, 512>>>(*this, res, kernel, width, height, kernel_width, kernel_height);
  hipDeviceSynchronize();


  return res;
}


Matrix Matrix::hypot(Matrix other)
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      double val1 = (*this)[i][j];
      double val2 = other[i][j];
      double r = sqrt(val1 * val1 + val2 * val2);

      res[i][j] = r;
    }
  }

  return res;
}


Matrix Matrix::arctan2(Matrix other)
{
  Matrix res(width, height);

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      double val1 = (*this)[i][j];
      double val2 = other[i][j];
      double r = atan(val1 / val2);

      res[i][j] = r;
    }
  }

  return res;
}
