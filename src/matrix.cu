#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>
#include "debug.h"
#include "matrix.h"

using namespace Pixor;

const int NUM_BLOCKS = 100;
const int THREADS_PER_BLOCK = 512;

// Try to mutate current matrix instead of returning new ones
// and calling hipMallocManaged too many times.

__host__
__device__
Row::Row(int length, double *r) :
  r(r),
  length(length)
  {}

__host__
__device__
double &Row::operator[](int index)
{
  assert(index >= 0 && index < length);
  return r[index];
}


Matrix::Matrix(int width, int height) :
  width(width),
  height(height)
{
  hipMallocManaged(&m, width * height * sizeof(double));

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      (*this)[i][j] = 0;
    }
  }
}


Matrix::Matrix(std::vector<std::vector<double>> matrix) :
  width(matrix[0].size()),
  height(matrix.size())
{
  hipMallocManaged(&m, width * height * sizeof(double));

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      (*this)[i][j] = matrix[i][j];
    }
  }
}

__host__
__device__
int Matrix::get_width()
{
  return width;
}

__host__
__device__
int Matrix::get_height()
{
  return height;
}

__host__
__device__
Row Matrix::operator[](int index)
{
  assert(index >= 0 && index < height);
  return Row(width, m + index * width);
}


void Matrix::print()
{
  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      std::cout << (*this)[i][j] << " ";
    }
    std::cout << '\n';
  }
}

__global__
void matrix_power(Matrix src, Matrix dest, int exponent)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = pow(src.m[index], exponent);
  }
}

Matrix Matrix::power(int exponent)
{
  Matrix res(width, height);

  matrix_power<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, exponent);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_add(Matrix src, Matrix dest, Matrix other)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = src.m[index] + other.m[index];
  }
}

Matrix Matrix::add(Matrix other)
{
  Matrix res(width, height);

  matrix_add<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, other);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_mult(Matrix src, Matrix dest, double k)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = src.m[index] * k;
  }
}

Matrix Matrix::mult(double k)
{
  Matrix res(width, height);

  matrix_mult<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, k);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_div(Matrix src, Matrix dest, double k)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = src.m[index] / k;
  }
}

Matrix Matrix::div(double k)
{
  Matrix res(width, height);

  matrix_div<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, k);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_exp(Matrix src, Matrix dest)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = std::exp(src.m[index]);
  }
}

Matrix Matrix::exp()
{
  Matrix res(width, height);

  matrix_exp<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_neg(Matrix src, Matrix dest)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    dest.m[index] = -src.m[index];
  }
}

Matrix Matrix::neg() {
  Matrix res(width, height);

  matrix_neg<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res);
  hipDeviceSynchronize();
  
  return res;
}

double Matrix::sum() {
  double res = 0;

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      res += (*this)[i][j];
    }
  }

  return res;
}


double Matrix::max() {
  double res = (*this)[0][0];

  for (int i = 0; i < height; i++) {
    for (int j = 0; j < width; j++) {
      double val = (*this)[i][j];
      if (val > res) res = val;
    }
  }

  return res;
}

__global__
void matrix_convolve(Matrix src, Matrix dest, Matrix kernel, int width, int height, int kernel_width, int kernel_height)
{
  int offset = (kernel_width - 1) / 2;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    int row = index / width;
    int col = index - row * width;
    double val = 0;

    for (int kernel_row = 0; kernel_row < kernel_height; kernel_row++) {
      for (int kernel_col = 0; kernel_col < kernel_width; kernel_col++) {
        int src_row = row + kernel_row - offset;
        int src_col = col + kernel_col - offset;
        if (src_row < 0 || src_row > height - 1) {
          src_row = row + (kernel_height - kernel_row) - offset;
        }
        if (src_col < 0 || src_col > width - 1) {
          src_col = col + (kernel_width - kernel_col) - offset;
        }
        double src_val = src[src_row][src_col];
        double k_val = kernel[kernel_height - 1 - kernel_row][kernel_width - 1 - kernel_col];

        val += k_val * src_val;
      }
    }

    dest[row][col] = val;
  }
}


Matrix Matrix::convolve(Matrix kernel) {
  Matrix res(width, height);
  int kernel_width = kernel.get_width();
  int kernel_height = kernel.get_height();
  assert(kernel_width == kernel_height);
  assert(kernel_width % 2 == 1);

  matrix_convolve<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, kernel, width, height, kernel_width, kernel_height);
  hipDeviceSynchronize();

  return res;
}

__global__
void matrix_hypot(Matrix src, Matrix dest, Matrix other)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    double val1 = src.m[index];
    double val2 = other.m[index];
    double r = sqrt(val1 * val1 + val2 * val2);

    dest.m[index] = r;
  }
}

Matrix Matrix::hypot(Matrix other)
{
  Matrix res(width, height);

  matrix_hypot<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, other);
  hipDeviceSynchronize();
  
  return res;
}

__global__
void matrix_arctan2(Matrix src, Matrix dest, Matrix other)
{
  int width = src.get_width();
  int height = src.get_height();
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    double val1 = src.m[index];
    double val2 = other.m[index];
    double r = atan(val1 / val2);

    dest.m[index] = r;
  }
}

Matrix Matrix::arctan2(Matrix other)
{
  Matrix res(width, height);

  matrix_arctan2<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(*this, res, other);
  hipDeviceSynchronize();
  
  return res;
}
