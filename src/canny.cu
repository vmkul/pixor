#include "hip/hip_runtime.h"
#include "canny.h"
#include "debug.h"
#include <cassert>
#include <math.h>

const int NUM_BLOCKS = 256;
const int THREADS_PER_BLOCK = 750;

Pixor::Matrix x_mgrid(int val)
{
  int size = val * 2 + 1;
  Pixor::Matrix res(size, size);

  for (int row = 0; row < size; row++) {
    for (int i = 0; i < size; i++) {
      res[row][i] = -val + i;
    }
  }

  return res;
}

Pixor::Matrix y_mgrid(int val)
{
  int size = val * 2 + 1;
  Pixor::Matrix res(size, size);

  for (int col = 0; col < size; col++) {
    for (int i = 0; i < size; i++) {
      res[i][col] = -val + i;
    }
  }

  return res;
}

Pixor::Matrix gaussian_kernel(int size, double sigma = 1)
{
  assert(size % 2 == 1);
  size /= 2;
  auto x = x_mgrid(size);
  auto y = y_mgrid(size);
  double normal = 1 / (2.0 * M_PI * pow(sigma, 2));

  auto kernel = x.power(2).add(y.power(2)).div(2.0 * pow(sigma, 2)).neg().exp().mult(normal);
  return kernel.div(kernel.sum());
}

Pixor::Matrix sobel_filter(Pixor::Matrix &m, Pixor::Matrix &theta)
{
  std::vector<std::vector<double>> kx_v = {{-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1}};
  std::vector<std::vector<double>> ky_v = {{1, 2, 1}, {0, 0, 0}, {-1, -2, -1}};
  Pixor::Matrix kx(kx_v);
  Pixor::Matrix ky(ky_v);

  auto ix = m.convolve(kx);
  auto iy = m.convolve(ky);
  auto hypot = ix.hypot(iy);
  hypot = hypot.div(hypot.max()).mult(255);
  theta = iy.arctan2(ix);

  return hypot;
}

__global__
void normalize_angle(Pixor::Matrix angle)
{
  int width = angle.width;
  int height = angle.height;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    auto val = angle.m[index];
    if (val < 0) {
      angle.m[index] += 180;
    }
  }

}

__device__
int get_index(int i, int j, int width) {
  return i * width + j;
}

__global__
void cuda_non_max_suppression(Pixor::Matrix angle, Pixor::Matrix m, Pixor::Matrix res)
{
  int width = angle.width;
  int height = angle.height;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
      double q = 255;
      double r = 255;
      auto angle_val = angle.m[index];
      int i = index / width;
      int j = index - i * width;

      if ((angle_val >= 0 && angle_val < 22.5) || (angle_val >= 157.5 && angle_val <= 180)) {
        if (j + 1 > width - 1 || j - 1 < 0) continue;
        q = m.m[get_index(i, j + 1, width)];
        r = m.m[get_index(i, j - 1, width)];
      } else if (angle_val >= 22.5 && angle_val < 67.5) {
        if (j + 1 > width - 1 || j - 1 < 0) continue;
        if (i + 1 > height - 1 || i - 1 < 0) continue;
        q = m.m[get_index(i + 1, j - 1, width)];
        r = m.m[get_index(i - 1, j + 1, width)];
      } else if (angle_val >= 67.5 && angle_val < 112.5) {
        if (i + 1 > height - 1 || i - 1 < 0) continue;
        q = m.m[get_index(i + 1, j, width)];
        r = m.m[get_index(i - 1, j, width)];
      } else if (angle_val >= 112.5 && angle_val < 157.5) {
        if (j + 1 > width - 1 || j - 1 < 0) continue;
        if (i + 1 > height - 1 || i - 1 < 0) continue;
        q = m.m[get_index(i - 1, j - 1, width)];
        r = m.m[get_index(i + 1, j + 1, width)];
      }

      if (m.m[index] >= q && m.m[index] >= r) {
        res.m[index] = m.m[index];
      } else {
        res.m[index] = 0;
      }
  }
}

Pixor::Matrix non_max_suppression(Pixor::Matrix &m, Pixor::Matrix &theta)
{
  int width = m.get_width();
  int height = m.get_height();
  Pixor::Matrix res(width, height);
  auto angle = theta.mult(180).div(M_PI);
  normalize_angle<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(angle);
  cuda_non_max_suppression<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(angle, m, res);
  hipDeviceSynchronize();

  return res;
}

__global__
void cuda_threshold(Pixor::Matrix src, Pixor::Matrix res, int weak, int strong, double low_threshold, double high_threshold)
{
  int width = src.width;
  int height = src.height;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    double m_val = src.m[index];

    if (m_val >= high_threshold) {
      res.m[index] = strong;
    } else if (m_val >= low_threshold) {
      res.m[index] = weak;
    }
  }
}

Pixor::Matrix threshold(Pixor::Matrix &m, double low_threshold_ratio = 0.05, double high_threshold_ratio = 0.09, int weak_pixel = 25, int strong_pixel = 255)
{
  auto high_threshold = m.max() * high_threshold_ratio;
  auto low_threshold = high_threshold * low_threshold_ratio;
  int width = m.get_width();
  int height = m.get_height();
  Pixor::Matrix res(width, height);

  cuda_threshold<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(m, res, weak_pixel, strong_pixel, low_threshold, high_threshold);
  hipDeviceSynchronize();

  return res;
}

__global__
void cuda_hysteresis(Pixor::Matrix m, int weak, int strong)
{
  int width = m.width;
  int height = m.height;
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  for (; index < width * height; index += stride) {
    auto m_val = m.m[index];
    int i = index / width;
    int j = index - i * width;
    if (m_val != weak) continue;

    for (int row_offset = 0; row_offset < 3; row_offset++) {
      for (int col_offset = 0; col_offset < 3; col_offset++) {
        int src_row = i + row_offset - 1;
        int src_col = j + col_offset - 1;
        if (src_row == i && src_col == j) continue;
        if (src_row < 0 || src_row > height - 1 || src_col < 0 || src_col > width - 1) {
          continue;
        }
        if (m.m[get_index(src_row, src_col, width)] == strong) {
          m.m[index] = strong;
          goto endpixel;
        }
      }
    }
    m.m[index] = 0;
    endpixel:;
  }
}

Pixor::Matrix hysteresis(Pixor::Matrix &m, int weak = 25, int strong = 255)
{
  int width = m.get_width();
  int height = m.get_height();

  cuda_hysteresis<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(m, weak, strong);
  hipDeviceSynchronize();

  return m;
}

Pixor::Matrix canny_edge_detector(Pixor::Matrix &m, double sigma, int kernel_size, double low_threshold, double high_threshold, int weak_pixel)
{
  auto res = m.convolve(gaussian_kernel(kernel_size, sigma));
  Pixor::Matrix theta(1, 1);
  res = sobel_filter(res, theta);
  res = non_max_suppression(res, theta);
  res = threshold(res, low_threshold, high_threshold, weak_pixel);
  res = hysteresis(res, weak_pixel);

  int err = hipGetLastError();
  if (err != 0) {
    dbgln("CUDA ERROR!: %d", err);
    return Pixor::Matrix(0, 0);
  }

  return res;
}
